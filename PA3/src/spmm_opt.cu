#include "hip/hip_runtime.h"
#include "spmm_opt.h"
#include "util.h"

__global__ void csr2coo_kernel(int *ptr, int *coo, int num_v)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_v) return;
    int begin = ptr[tid], end = ptr[tid + 1];
    for (int i = begin; i < end; ++i)
    {
        coo[i] = tid;
    }
}

__global__ void spmm_kernel_opt(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE, int *coo, int num_e)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_e) return;
    int row = coo[tid];
    int col = idx[tid];
    float value = val[tid];
    for (int j = 0; j < INFEATURE; ++j)
    {
        atomicAdd(&vout[row * INFEATURE + j], vin[col * INFEATURE + j] * value);
    }
}

void SpMMOpt::preprocess(float *vin, float *vout)
{
    // Convert CSR to COO
    checkCudaErrors(cudaMalloc2((void**)&d_coo, num_e * sizeof(int)));
    csr2coo_kernel<<<(num_e + 127) / 128, 128>>>(d_ptr, d_coo, num_v);

    int BLOCK_SIZE = 128;
    grid.x = (num_e + BLOCK_SIZE - 1) / BLOCK_SIZE;
    block.x = BLOCK_SIZE;
}

void SpMMOpt::run(float *vin, float *vout)
{
    // TODO: your code
    spmm_kernel_opt<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, num_v, feat_in, d_coo, num_e);
}