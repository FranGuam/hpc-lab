#include "hip/hip_runtime.h"
#include "spmm_opt.h"
#include "util.h"
#include <vector>

#define ROW_NUM_32 1
#define ROW_THREAD_32 32
#define ROW_ELEM_32 64
#define ROW_NUM_256 1
#define ROW_THREAD_256 128
#define ROW_ELEM_256 128

__global__ void spmm_kernel_opt32(int *ptr, int *idx, float *val, float *vin, float *vout, int *iperm)
{
    __shared__ int s_idx[ROW_NUM_32 * ROW_ELEM_32];
    __shared__ float s_val[ROW_NUM_32 * ROW_ELEM_32];
    // int tid = blockIdx.x * blockDim.y + threadIdx.y;
    // if (tid >= num_v) return;
    int begin = ptr[blockIdx.x];
    int end = ptr[blockIdx.x + 1];
    int bound = end - threadIdx.x;
    // int offset = threadIdx.y * ROW_ELEM_32;
    // int *s_idx_base = s_idx + offset;
    // float *s_val_base = s_val + offset;
    int *s_idx_base = s_idx + threadIdx.x;
    float *s_val_base = s_val + threadIdx.x;
    float *vin_base = vin + threadIdx.x;
    int *idx_base = idx + threadIdx.x;
    float *val_base = val + threadIdx.x;

    float tmp = 0;
    for (int i = begin; i < end; i += ROW_ELEM_32)
    {
        // Load data into shared memory
        if (i < bound)
        {
            *s_idx_base = idx_base[i];
            *s_val_base = val_base[i];
        }
        int ii = i + ROW_THREAD_32;
        if (ii < bound)
        {
            s_idx_base[ROW_THREAD_32] = idx_base[ii];
            s_val_base[ROW_THREAD_32] = val_base[ii];
        }
        __syncwarp();

        // Compute
        int max = min(ROW_ELEM_32, end - i);
        for (int j = 0; j < max; ++j)
        {
            tmp += vin_base[(s_idx[j] << 5)] * s_val[j];
        }
        __syncwarp();
    }
    // vout[(tid << 5) + threadIdx.x] = tmp;
    vout[(iperm[blockIdx.x] << 5) + threadIdx.x] = tmp;
}

__global__ void spmm_kernel_opt256(int *ptr, int *idx, float *val, float *vin, float *vout, int *iperm)
{
    __shared__ int s_idx[ROW_NUM_256 * ROW_ELEM_256];
    __shared__ float s_val[ROW_NUM_256 * ROW_ELEM_256];
    // int tid = blockIdx.x * blockDim.y + threadIdx.y;
    // if (tid >= num_v) return;
    // int begin = ptr[tid], end = ptr[tid + 1];
    int begin = ptr[blockIdx.x];
    int end = ptr[blockIdx.x + 1];
    int bound = end - threadIdx.x;
    // int offset = threadIdx.y * ROW_ELEM_256;
    // int *s_idx_base = s_idx + offset;
    // float *s_val_base = s_val + offset;
    int *s_idx_base = s_idx + threadIdx.x;
    float *s_val_base = s_val + threadIdx.x;
    float *vin_base1 = vin + threadIdx.x;
    float *vin_base2 = vin_base1 + ROW_THREAD_256;
    int *idx_base = idx + threadIdx.x;
    float *val_base = val + threadIdx.x;

    float tmp1 = 0, tmp2 = 0;
    for (int i = begin; i < end; i += ROW_ELEM_256)
    {
        // Load data into shared memory
        if (i < bound)
        {
            *s_idx_base = idx_base[i];
            *s_val_base = val_base[i];
        }
        __syncthreads();

        // Compute
        int max = min(ROW_ELEM_256, end - i);
        for (int j = 0; j < max; ++j)
        {
            int tmp_idx = s_idx[j] << 8;
            float tmp_val = s_val[j];
            tmp1 += vin_base1[tmp_idx] * tmp_val;
            tmp2 += vin_base2[tmp_idx] * tmp_val;
        }
        __syncthreads();
    }
    // vout[(tid << 8) + threadIdx.x] = tmp1;
    // vout[(tid << 8) + threadIdx.x + ROW_THREAD_256] = tmp2;
    vout += (iperm[blockIdx.x] << 8) + threadIdx.x;
    *vout = tmp1;
    vout[ROW_THREAD_256] = tmp2;
}

void SpMMOpt::preprocess(float *vin, float *vout)
{
    if (feat_in == 32)
    {
        block.y = ROW_NUM_32;
        // grid.x = (num_v + block.y - 1) / block.y;
        grid.x = num_v;
        block.x = ROW_THREAD_32;
    }
    else
    {
        block.y = ROW_NUM_256;
        // grid.x = (num_v + block.y - 1) / block.y;
        grid.x = num_v;
        block.x = ROW_THREAD_256;
    }

    // Copy data to host
    std::vector<int> row_ptr(num_v + 1);
    std::vector<int> col_idx(num_e);
    std::vector<float> val(num_e);
    checkCudaErrors(hipMemcpy(row_ptr.data(), d_ptr, sizeof(int) * (num_v + 1), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(col_idx.data(), d_idx, sizeof(int) * num_e, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(val.data(), d_val, sizeof(float) * num_e, hipMemcpyDeviceToHost));
    // Re-order the rows of the matrix
    std::vector<std::vector<int>> columns(num_v);
    for (int row = 0; row < num_v; ++row) {
        for (int idx = row_ptr[row]; idx < row_ptr[row + 1]; ++idx) {
            columns[col_idx[idx]].push_back(row);
        }
    }
    std::vector<int> perm(num_v, -1);
    std::vector<int> iperm(num_v, -1);
    int new_row = 0;
    for (int col = 0; col < num_v; ++col) {
        for (int row : columns[col]) {
            if (perm[row] == -1) {
                perm[row] = new_row++;
                iperm[perm[row]] = row;
            }
        }
    }
    // Create a new matrix
    std::vector<int> new_row_ptr(num_v + 1);
    std::vector<int> new_col_idx(num_e);
    std::vector<float> new_val(num_e);
    std::vector<int> new_row_len(num_v);
    for (int row = 0; row < num_v; ++row) {
        int new_row = perm[row];
        new_row_len[new_row] = row_ptr[row + 1] - row_ptr[row];
    }
    new_row_ptr[0] = 0;
    for (int new_row = 0; new_row < num_v; ++new_row) {
        new_row_ptr[new_row + 1] = new_row_ptr[new_row] + new_row_len[new_row];
    }
    for (int row = 0; row < matrix.num_rows; ++row) {
        int new_row = perm[row];
        int new_idx = new_row_ptr[new_row];
        for (int idx = row_ptr[row]; idx < row_ptr[row + 1]; ++idx, ++new_idx) {
            new_col_idx[new_idx] = col_idx[idx];
            new_val[new_idx] = val[idx];
        }
    }
    // Copy data back to device
    checkCudaErrors(hipMemcpy(d_ptr, new_row_ptr.data(), sizeof(int) * (num_v + 1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_idx, new_col_idx.data(), sizeof(int) * num_e, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_val, new_val.data(), sizeof(float) * num_e, hipMemcpyHostToDevice));
    checkCudaErrors(cudaMalloc2(&d_iperm, sizeof(int) * num_v));
    checkCudaErrors(hipMemcpy(d_iperm, iperm.data(), sizeof(int) * num_v, hipMemcpyHostToDevice));
}

void SpMMOpt::run(float *vin, float *vout)
{
    if (feat_in == 32)
    {
        spmm_kernel_opt32<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, d_iperm);
    }
    else
    {
        spmm_kernel_opt256<<<grid, block>>>(d_ptr, d_idx, d_val, vin, vout, d_iperm);
    }
}
